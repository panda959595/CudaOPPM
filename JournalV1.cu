#include "hip/hip_runtime.h"
//!매치 정보는 순위동형이 발생하는 텍스트에서의 위치만 전달한다!

//더 효율적으로 짤수 있지만 병렬화 전의 논문과의 비교를 위해 다른 인자는 통일해야 한다고 생각, 효율화 하지 않음
//검색단계만 병렬적으로 효율화를 함
//하지만 기존의 preprocessing_phi()는 너무 비효율적으로 작동하여 수정함, 대응되는게 MakeTempLoc()

#define _CRT_SECURE_NO_WARNINGS

#include ""
#include "cuda_by_example/common/book.h"
#include "hip/hip_runtime.h"
#include ""
#include<sys/time.h>
#include<cstdlib>
#include<stdio.h>
#include<fstream>
#include<cstring>

//Merge Sort에서 사용하는 값. 패턴의 길이를 넘어가지 않음
#define Repeat 5
#define MAX_COUNT 1'000
#define ThreadCount 1'024
#define CopySize 1'000'005
#define GpuTextLen 100
using namespace std;

typedef pair<int,int> P;

__constant__ int DevLoc[16'000]; //MAX
__constant__ int DevPreCalFac[10];

//Input Folder Name
string InputFolder = "./TESTCASE/TC-";
string OutputFolder = "./JournalV1OUTPUT/TC-";
string TimeFolder = "./JournalV1TIME/";
string TextInput = "TextSample";
string PatternInput = "IntStr";
string TimeInput = "TimeRecord_";

struct timeval PreStart, PreEnd, SearchStart, SearchEnd, TotalStart, TotalEnd, CopyToHostStart, CopyToHostEnd;

int PreCalFac[10] = { 0, 1, 2, 6, 24, 120, 720, 5040, 40320, 362880}; //0!~9!

void InputData(int ** Pattern, int * Text, int PatternCount, int PatternLen, int TextLen, int FolderNumber){
	//Pattern input
	string pattern_filename = InputFolder + to_string(FolderNumber)+"/"+PatternInput + "_" + to_string(PatternCount) + "_" +to_string(PatternLen) + ".txt";
	ifstream pattern(pattern_filename);
	
	for (int i = 0; i < PatternCount; i++) {
    	for (int j = 0; j < PatternLen; j++) {
        	pattern >> Pattern[i][j];
		}
	}
	pattern.close();

	//Text input
	string text_filename = InputFolder + to_string(FolderNumber)+"/"+ TextInput + "_" + to_string(TextLen) + ".txt";
	ifstream text(text_filename);

	for (int i = 0; i < TextLen; i++) {
		text >> Text[i];
	}
	text.close();
	return ;
}

void OutputData(int PatternCount, int PatternLen, int TextLen,int BlockSize, int FolderNumber,int MatchRes, bool * MatchResDetail){
	string FileName = OutputFolder+ to_string(FolderNumber)+"/"+PatternInput + "_" +
	 to_string(PatternCount) + "_" +to_string(PatternLen) +"_"+to_string(TextLen) +"_"+to_string(BlockSize) + ".txt";
	 
	ofstream FileStream(FileName);
	FileStream<<MatchRes;
	/*FileStream<<"\n";
	for(int t=0;t<TextLen; t++){
		FileStream<<MatchResDetail[t]<<" ";
	}*/
	FileStream.close();
}

void OutputTime(double Pre, float Search, double Total,double TotalCopy, int PatternCount,int PatternLen, int TextLen,int BlockSize){
	string FileName = TimeFolder + PatternInput + "_" +
					  to_string(PatternCount) + "_" + to_string(PatternLen) + "_" +
					   to_string(TextLen) + "_" + to_string(BlockSize)+".txt";

	ofstream FileStream(FileName);
	FileStream<<(double)(Pre)/Repeat<<" "<<(double)(Search)/Repeat<<" "
	<<(double)(Total)/Repeat<<" "<<(double)(TotalCopy)/Repeat;

	FileStream.close();
}

ofstream GetFileStream(int PatternCount, int PatternLen){
	string FileName = OutputFolder + "FP_" + to_string(PatternCount) + "_" + to_string(PatternLen) + ".txt";
	ofstream FileStream(FileName);
	return FileStream;
}

int FindLen(int* p, int PatternLen) {

	int ret = PatternLen;
	for (int i = 0; i < PatternLen; i++) {
		if (p[i] < 0 || p[i] == 0) {
			ret = i;
			break;
		}
	}

	return ret;
}

void merge(int first, int mid, int last, P* arr) {

	int idx = first;
	P TempArr[MAX_COUNT];


	int i = first, j = mid + 1;

	while (i <= mid && j <= last) {
		if (arr[i] <= arr[j]) {
			TempArr[idx] = arr[i];
			idx++;
			i++;
		}
		else if (arr[i] > arr[j]) {
			TempArr[idx] = arr[j];
			idx++;
			j++;
		}
	}

	if (i > mid) {
		for (int m = j; m <= last; m++) {
			TempArr[idx] = arr[m];
			idx++;
		}
	}
	else {
		for (int m = i; m <= mid; m++) {
			TempArr[idx] = arr[m];
			idx++;
		}
	}

	for (int m = first; m <= last; m++) {
		arr[m] = TempArr[m];
	}
}

void mergeSort(int first, int last, P* TempPattern) {

	if (first < last) {
		int mid = (first + last) / 2;
		mergeSort(first, mid, TempPattern);
		mergeSort(mid + 1, last, TempPattern);
		merge(first, mid, last, TempPattern);
	}
}

int FindMax(int* p, int len) {
	int ret = 0;
	for (int i = 0; i < len; i++) {
		if (p[i] > ret)
			ret = p[i];
	}
	return ret;
}

int CalQgram(int* Pattern, int StartIdx, int PatternLen, int BlockSize) {
	int result = 0;
	int count;

	for (int j = StartIdx; j < StartIdx + BlockSize; j++) {
		count = 0;
		for (int k = StartIdx; k < j; k++) {
			if (Pattern[k] <= Pattern[j])
				count++;
		}
		result += count * PreCalFac[j-StartIdx];
	}
	return result;
}

__device__ int DevCalQgram(int Text[], int StartIdx, int PatternLen, int BlockSize){
	int result = 0;
	int count;

	for (int j = StartIdx; j < StartIdx + BlockSize; j++) {
		count = 0;
		for (int k = StartIdx; k < j; k++) {
			if (Text[k] <= Text[j])
				count++;
		}
		result += count * DevPreCalFac[j - StartIdx];
	}
	return result;
}

//Len과 PatternLen은 중복되는 정보이나 본 알고리즘에서
//패턴의 길이가 전부다 다른 경우도 고려할 수 있도록 Len 변수는 남겨둠.
//Loc table은 가로 * 세로 => 패턴길이 * 패턴개수인 논리적으로는 2차원이지만 실제로는 1차원인 배열임
void MakeLoc(P* TempPattern, int* Loc, int Len, int PatternCount,int PatternLen, int CurPatternIdx) {
	for (int i = 0; i < Len; i++) {
		int idx = CurPatternIdx + i * PatternCount;
		Loc[idx] = TempPattern[i].second;
	}
}

void MakeE(int* Pattern, int* Loc, int* E, int Len,int PatternCount, int CurPatternIdx) {
	for (int i = 0; i < Len - 1; i++) {
		int idx = CurPatternIdx + i * PatternCount;

		if (Pattern[Loc[idx]] == Pattern[Loc[idx + PatternCount]])
			E[idx] = 1;
		else
			E[idx] = 0;
	}
}

void FillLoc(int ** Pattern, int * Loc, int* E, int PatternCount, int PatternLen){
	int Len;
	P* TempPattern;

	for (int i = 0; i < PatternCount; i++) {
		Len = FindLen(Pattern[i], PatternLen);
		TempPattern = new P[Len];

		for (int j = 0; j < Len; j++) {
			TempPattern[j].first = Pattern[i][j];
			TempPattern[j].second = j;
		}
		mergeSort(0, Len - 1, TempPattern);
				
		MakeLoc(TempPattern, Loc, Len, PatternCount, PatternLen, i);

		MakeE(Pattern[i], Loc, E, Len, PatternCount, i);
		delete[] TempPattern;
	}
}

void FillHash(int **Pattern, int BlockSize, int PatternCount, int PatternLen, int * Hash){
	int range = PatternLen - BlockSize + 1;

	for (int i = 0; i < PatternCount; i++) {
		Hash[i] = CalQgram(Pattern[i], range - 1, PatternLen, BlockSize);
	}
}

//__device__ InitSharedMemory()
__device__ bool CheckOP(int Text[], int* E, int StartIdx, int PatternLen, int PatternIdx, int PatternCount) {
	
	bool ret = true;
	for (int i = 0; i < PatternLen-1; i++) {
		int idx = PatternCount * i + PatternIdx;
		
		if (E[idx] == 0) {
			if (Text[StartIdx + DevLoc[idx]] >= Text[StartIdx + DevLoc[idx + PatternCount]]) {
				ret = false;
				break;
			}
		}

		else {
			if (Text[StartIdx + DevLoc[idx]] != Text[StartIdx + DevLoc[idx + PatternCount]]) {
				ret = false;
				break;
			}
		}
	}
	return ret;
}


__global__ void Search(int * DevText, int * DevHash,int * DevE,int * DevMatchRes,
	 int TextLen, int PatternCount, int PatternLen,int BlockSize,bool * DevMatchDetail){

	extern __shared__ int sharedText[]; //dynamic allocation
	int bidx = blockIdx.x;
	int tidx = threadIdx.x;
	int TextRange = GpuTextLen + PatternLen;
	int TextStart = bidx * GpuTextLen;

	//마지막 block일때 길이.
	int CurTextLen = (TextLen/GpuTextLen) -1 == bidx ? GpuTextLen-PatternLen : GpuTextLen;

	if(tidx<TextRange && (TextStart + tidx < TextLen)){
		sharedText[tidx] = DevText[TextStart+tidx];
	}
	__syncthreads();
	if(tidx<PatternCount){
		for(int i=0; i < CurTextLen; i++){
			int temp = DevCalQgram(sharedText, i+PatternLen-BlockSize, PatternLen, BlockSize);
			
			if(temp == DevHash[tidx]){
				if(CheckOP(sharedText, DevE, i,PatternLen, tidx, PatternCount)){
				//atomicAdd(&DevMatchRes[0], 1);
				DevMatchDetail[(TextStart+i) + (tidx * TextLen)] = true;
				}
			}
		}
	}
	__syncthreads();
}

extern "C" void InitLocGpu(int * Loc,int PatternCount, int PatternLen)
{
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(DevLoc), Loc, PatternCount * PatternLen * sizeof(int)));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(DevPreCalFac), PreCalFac, 10 * sizeof(int)));
}

void FreeVariable(int * DevMatchRes,int * DevHash,int * DevText, int *DevE,
	int * Text, int **Pattern,int * Loc,int * Hash,int * E, int PatternCount,int * MatchRes, bool *MatchResDetail, bool * DevMatchDetail){
	
	for(int i=0;i<PatternCount;i++){
		delete[] Pattern[i];
	}
	delete[] Text;
	delete[] Loc;
	delete[] Hash;
	delete[] E;
	delete[] MatchRes;
	delete[] MatchResDetail;
	hipFree(DevE);
	hipFree(DevMatchRes);
	hipFree(DevHash);
	hipFree(DevText);
	hipFree(DevMatchDetail);
}

void PrintTestInfo(int PatternCount,int PatternLen,int TextLen, int MatchRes){
	printf("Pattern count: %d Pattern_length : %d TEXT SIZE : %d\nOP size : %d\n\n", PatternCount, PatternLen,TextLen, MatchRes);
}

int main(){
	int ** Pattern;
	int * Loc;
	int * E;
	int * Hash;
	int * Text;
	int * MatchRes;
	bool * MatchResDetail;

	//GPU variables
	int * DevMatchRes;
	int * DevHash;
	int * DevText;
	int * DevE;
	bool * DevMatchDetail;

	for (int BlockSize = 7; BlockSize <= 7; BlockSize++) {
		for (int PatternCount = 100; PatternCount <= 1'000; PatternCount += 100) { // 100~1000
			for (int PatternLen = 7; PatternLen <= 15; PatternLen += 1) { //3~15
				printf("Pattern Count: %d\nPattern Len : %d\n",PatternCount, PatternLen);

				for (int TextLen = 100'000; TextLen <= 1'000'000; TextLen += 100'000) { //100'000 ~ 1'000'000
				double sec, usec;
				double TotalPre = 0;
				double TotalSearch = 0;
				double Total = 0;
				double TotalCopy = 0;
				for(int FolderNumber = 0;FolderNumber < Repeat;FolderNumber++){
					Text = new int[TextLen];

					//!Warning! Only this two table is row * col => PatternLen * PatternCount
					Loc = new int[PatternLen * PatternCount];
					E = new int[PatternLen * PatternCount];
					Hash = new int[PatternCount];

					Pattern = new int*[PatternCount];
					for (int i = 0; i < PatternCount; i++) {
						Pattern[i] = new int[PatternLen];
					}
					MatchResDetail = new bool[TextLen * PatternCount];

					//Read Text and Pattern
					InputData(Pattern, Text, PatternCount, PatternLen, TextLen,FolderNumber);

					gettimeofday(&TotalStart, NULL);

					//Fill the Location table
					gettimeofday(&PreStart, NULL);
					FillLoc(Pattern, Loc, E, PatternCount, PatternLen);

					//Fill the hash table
					FillHash(Pattern, BlockSize, PatternCount, PatternLen, Hash);
					gettimeofday(&PreEnd, NULL);

					//GPU Init !InitLocGpu는 관리자 권한으로 실행해야함!
					InitLocGpu(Loc, PatternCount, PatternLen);
					
					//GPU init
					HANDLE_ERROR(hipMalloc((void**)&DevMatchRes, sizeof(int) * 1));
					HANDLE_ERROR(hipMalloc((void**)&DevHash, sizeof(int) * PatternCount));
					HANDLE_ERROR(hipMalloc((void**)&DevText, sizeof(int) * TextLen));
					HANDLE_ERROR(hipMalloc((void**)&DevE, sizeof(int) * PatternCount * PatternLen));
					HANDLE_ERROR(hipMalloc((void**)&DevMatchDetail, TextLen*PatternCount * sizeof(bool)));

					HANDLE_ERROR(hipMemcpy(DevHash, Hash, sizeof(int) * PatternCount, hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(DevText, Text, sizeof(int) * TextLen, hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(DevE, E, sizeof(int) * PatternCount * PatternLen, hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemset(DevMatchRes, 0, sizeof(int)));
					HANDLE_ERROR(hipMemset(DevMatchDetail, 0 ,TextLen*PatternCount*sizeof(bool)));

					//Kernel !3rd parameter is shared memory size in byte. Take care!
					gettimeofday(&SearchStart, NULL);
					//블럭개수 늘리기
					Search<<<(TextLen/GpuTextLen), ThreadCount, 1000>>>(DevText, DevHash, DevE, DevMatchRes, TextLen, PatternCount, PatternLen,BlockSize,DevMatchDetail);
					hipDeviceSynchronize();

					gettimeofday(&SearchEnd, NULL);
					
					MatchRes = new int[2];
					gettimeofday(&CopyToHostStart,NULL);
					HANDLE_ERROR(hipMemcpy(MatchResDetail, DevMatchDetail, sizeof(bool) * TextLen * PatternCount, hipMemcpyDeviceToHost));
					HANDLE_ERROR(hipMemcpy(MatchRes, DevMatchRes, sizeof(int), hipMemcpyDeviceToHost));
					gettimeofday(&CopyToHostEnd,NULL);
					
					//PrintTestInfo(PatternCount, PatternLen,TextLen, MatchRes[0]);
					OutputData(PatternCount, PatternLen, TextLen, BlockSize,FolderNumber, MatchRes[0], MatchResDetail);
					//Freeing Variable
					FreeVariable(DevMatchRes, DevHash, DevText,DevE, Text, Pattern, Loc, Hash, E, PatternCount, MatchRes, MatchResDetail, DevMatchDetail);
					gettimeofday(&TotalEnd, NULL);
					
					sec = TotalEnd.tv_sec - TotalStart.tv_sec;
					usec = TotalEnd.tv_usec - TotalStart.tv_usec;
					Total += (sec*1000+usec/1000.0);

					sec = PreEnd.tv_sec - PreStart.tv_sec;
					usec = PreEnd.tv_usec - PreStart.tv_usec;
					TotalPre += (sec*1000+usec/1000.0);

					sec = SearchEnd.tv_sec - SearchStart.tv_sec;
					usec = SearchEnd.tv_usec - SearchStart.tv_usec;
					TotalSearch += (sec*1000+usec/1000.0);

					sec = CopyToHostEnd.tv_sec - CopyToHostStart.tv_sec;
					usec = CopyToHostEnd.tv_usec - CopyToHostStart.tv_usec;
					TotalCopy += (sec*1000+usec/1000.0); 
				}	
				//Folder End
				OutputTime(TotalPre, TotalSearch, Total,TotalCopy,PatternCount,PatternLen, TextLen,BlockSize);
			}
		}
	}
}
	return 0;
}