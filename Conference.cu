#include "hip/hip_runtime.h"
/*

2020.01.02

검색단계 병렬화 correctness 확인완료

*/

#define _CRT_SECURE_NO_WARNINGS

#include<iostream>

#include<ctime>

#include<fstream>

#include<cstdlib>

#include<iomanip>

#include<cmath>

#include<string>

#include "cuda_by_example/common/book.h"

#include "hip/hip_runtime.h"

#include ""

 

using namespace std;

 

#define MAX_COUNT 2000
#define MAX_LEN 20
#define CopySize 1'000'000
//#define BlockSize 3

//#define TextLen 1048575

 

#define min(a,b) a<b?a:b

#define max(a,b) a<b?b:a

string InputFolder = "./TESTCASE/TC-";
string OutputFolder = "./OUTPUT/TC-";
string TimeFolder = "./TIME/";
string TextInput = "TextSample";
string PatternInput = "IntStr";
string TimeInput = "TimeRecord_";
clock_t CopyToHostStart;
clock_t CopyToHostEnd;
clock_t SearchStart;
clock_t SearchEnd;
clock_t TotalStart;
clock_t TotalEnd;



class Hash_T {

public:

	int* pattern;

	Hash_T* next;

	int p_num;

	Hash_T();

};

Hash_T::Hash_T() {

 

	this->pattern = NULL;

	this->next = NULL;

	this->p_num = -1;

}

struct inv_H {

	int pattern_num;

	int FP_value;

};

 

int** make_p_prime(int** p, int m, int PatternCount) {

 

	int** temp = new int*[PatternCount];

	for (int i = 0; i < PatternCount; i++) {

		temp[i] = new int[m];

	}

 

	for (int i = 0; i < PatternCount; i++) {

		for (int j = 0; j < m; j++) {

			temp[i][j] = p[i][j];

		}

	}

 

	return temp;

}

void merge_H(int first, int mid, int last, inv_H* arr) {

 

	int idx = first;

 

	inv_H temp_arr[MAX_COUNT];

	int i = first, j = mid + 1;

 

	while (i <= mid && j <= last) {

		if (arr[i].FP_value <= arr[j].FP_value) {

			temp_arr[idx] = arr[i];

			idx++;

			i++;

		}

		else if (arr[i].FP_value > arr[j].FP_value) {

			temp_arr[idx] = arr[j];

			idx++;

			j++;

		}

	}

 

	if (i > mid) {

		for (int m = j; m <= last; m++) {

			temp_arr[idx] = arr[m];

			idx++;

		}

	}

	else {

		for (int m = i; m <= mid; m++) {

			temp_arr[idx] = arr[m];

			idx++;

		}

	}

 

	for (int m = first; m <= last; m++) {

		arr[m] = temp_arr[m];

	}

}

void mergeSort_H(int first, int last, inv_H* arr) {

	if (first < last) {

		int mid = (first + last) / 2;

		mergeSort_H(first, mid, arr);

		mergeSort_H(mid + 1, last, arr);

		merge_H(first, mid, last, arr);

	}

}

int Binary_Search(inv_H* arr, int size, int find_FP) {

 

	int low = 0, high = size - 1, mid;

 

	while (low <= high) {

		mid = (low + high) / 2;

		if (arr[mid].FP_value > find_FP) {

			high = mid - 1;

		}

		else if (arr[mid].FP_value < find_FP) {

			low = mid + 1;

		}

		else {

			return mid;

		}

	}

 

	return -1;

}

int factorial(int n) {

	return (n == 1 || n == 0) ? 1 : factorial(n - 1)*n;

}

 

__device__ int fac(int n) {

	return (n == 1 || n == 0) ? 1 : fac(n - 1)*n;

}

int q_gram_H(int* p, int s_idx, int m, int q) {

 

	//int size = m - q + 1;

	//int* ret = new int[size];

	int ret;

	int result;

	int count;

 

	//for (int i = 0; i < size; i++) {

 

	int i = s_idx;

	result = 0;

 

	for (int j = i; j < i + q; j++) {

		count = 0;

		for (int k = i; k < j; k++) {

			if (p[k] <= p[j])

				count++;

		}

		result += count * factorial(j - i);

	}

 

	ret = result;

	//}

	return ret;

}

 

__device__ int q_gram(int* p, int s_idx, int m, int q) {

 

	//int size = m - q + 1;

	//int* ret = new int[size];

	int ret;

	int result;

	int count;

 

	//for (int i = 0; i < size; i++) {

 

	int i = s_idx;

	result = 0;

 

	for (int j = i; j < i + q; j++) {

		count = 0;

		for (int k = i; k < j; k++) {

			if (p[k] <= p[j])

				count++;

		}

		result += count * fac(j - i);

	}

 

	ret = result;

	//}

	return ret;

}

void preprocessing_table(int** p, int B_size, int PatternCount, int PatternLen, int* Hash_Arr, inv_H* inverse_Hash) {

 

	int m = PatternLen;

	int range = m - B_size + 1;

	int** p_prime = make_p_prime(p, m, PatternCount);

 

	for (int i = 0; i < PatternCount; i++) {

		Hash_Arr[i] = q_gram_H(p_prime[i], range - 1, m, B_size);

	}

	for (int i = 0; i < PatternCount; i++)

		delete[] p_prime[i];

	delete[] p_prime;

}

 

int find_len_H(int* p, int PatternLen) {

 

	int ret = PatternLen;

	for (int i = 0; i < PatternLen; i++) {

		if (p[i] < 0 || p[i] == 0) {

			ret = i;

			break;

		}

	}

 

	return ret;

}

 

__device__ int find_len(int* p, int arr_idx, int PatternLen) {

 

	int ret = PatternLen;

	for (int i = arr_idx; i < PatternLen + arr_idx; i++) {

		if (p[i] < 0 || p[i] == 0) {

			ret = i;

			break;

		}

	}

	return ret;

}

__device__ int find_max_H(int* p,int s_idx, int len) {

 

	int ret = 0;

 

	for (int i = s_idx; i < s_idx+len; i++) {

		if (p[i] > ret)

			ret = p[i];

	}

 

	return ret;

}

int find_max(int* p, int len) {

 

	int ret = 0;

 

	for (int i = 0; i < len; i++) {

		if (p[i] > ret)

			ret = p[i];

	}

 

	return ret;

}

void merge(int first, int mid, int last, int* arr) {

 

	int idx = first;

	int temp_arr[MAX_COUNT];

 

 

	int i = first, j = mid + 1;

 

	while (i <= mid && j <= last) {

		if (arr[i] <= arr[j]) {

			temp_arr[idx] = arr[i];

			idx++;

			i++;

		}

		else if (arr[i] > arr[j]) {

			temp_arr[idx] = arr[j];

			idx++;

			j++;

		}

	}

 

	if (i > mid) {

		for (int m = j; m <= last; m++) {

			temp_arr[idx] = arr[m];

			idx++;

		}

	}

	else {

		for (int m = i; m <= mid; m++) {

			temp_arr[idx] = arr[m];

			idx++;

		}

	}

 

	for (int m = first; m <= last; m++) {

		arr[m] = temp_arr[m];

	}

}

void mergeSort(int first, int last, int* arr) {

 

	if (first < last) {

		int mid = (first + last) / 2;

		mergeSort(first, mid, arr);

		mergeSort(mid + 1, last, arr);

		merge(first, mid, last, arr);

	}

}

 

void make_phi_inv(int* phi, int* phi_inv, int len) {

 

	for (int i = 0; i < len; i++) {

		phi_inv[phi[i]] = i;

	}

}

void make_E(int* p, int* phi_inv, int* E, int len) {

 

	for (int i = 0; i < len - 1; i++) {

		if (p[phi_inv[i]] == p[phi_inv[i + 1]])

			E[i] = 1;

		else

			E[i] = 0;

	}

}

__global__ void make_phi(int* temp_p_1d, int* p_1d, int* phi_1d, int len, int PatternCount) {

	//temp_p_1d 가 정렬되어있는것임

	//하나의 스레드가 하나의 파이 만든다

	int idx = blockDim.x*blockIdx.x + threadIdx.x;

	int arr_idx = idx * len;

 

	if (idx <PatternCount) {

		int * flag = new int[len];

		for (int i = 0; i < len; i++) {

			flag[i] = 0;

		}

		for (int i = arr_idx; i < arr_idx + len; i++) {

			phi_1d[i] = -1;

		}

 

		int temp;

		for (int i = arr_idx; i < arr_idx+len; i++) {

			temp = p_1d[i];

			for (int j = arr_idx; j < arr_idx+len; j++) {

				if (temp_p_1d[j] == temp && flag[j-arr_idx] == 0) {// temp_p_1d는 정렬되어 있음 

					phi_1d[i] = j - arr_idx;

					flag[j-arr_idx] = 1;

					break;

				}

			}

		}

		delete flag;

	}

}

 

void preprocessing_phi(int* pattern_1d, int** p, int** phi, int** phi_inv, int** E, int PatternCount, int PatternLen) {

	//string FOLDER = "./OUTPUT/";

	//string FILENAME = "out.txt";

	int* temp_pattern_1d = new int[PatternCount * PatternLen];

	int* temp_arr = new int[PatternLen];

	int* phi_1d = new int[PatternCount *PatternLen];

 

	//global 함수//

	int* dev_pattern_1d;

	int* dev_temp_pattern_1d;

	int* dev_phi_1d;

	//gpu 메모리 할당//

	HANDLE_ERROR(hipMalloc((void**)&dev_pattern_1d, PatternCount * PatternLen * sizeof(int)));

	HANDLE_ERROR(hipMalloc((void**)&dev_temp_pattern_1d, PatternCount * PatternLen * sizeof(int)));

	HANDLE_ERROR(hipMalloc((void**)&dev_phi_1d, PatternCount * PatternLen * sizeof(int)));

 

	for (int i = 0; i < PatternCount; i++) {

		for (int j = 0; j < PatternLen; j++) {

			temp_arr[j] = pattern_1d[i*PatternLen+j];

		}

		mergeSort(0, PatternLen - 1, temp_arr);

 

		for (int j = 0; j < PatternLen; j++) {

			temp_pattern_1d[i*PatternLen + j] = temp_arr[j];

		}

	}

 

	HANDLE_ERROR(hipMemcpy(dev_pattern_1d, pattern_1d, PatternCount * PatternLen * sizeof(int), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(dev_temp_pattern_1d, temp_pattern_1d, PatternCount * PatternLen * sizeof(int),hipMemcpyHostToDevice));

 

	make_phi << <(PatternCount + 127) / 128, 128 >> > (dev_temp_pattern_1d, dev_pattern_1d, dev_phi_1d, PatternLen, PatternCount);

	hipDeviceSynchronize();

	HANDLE_ERROR(hipMemcpy(phi_1d, dev_phi_1d, PatternCount * PatternLen * sizeof(int), hipMemcpyDeviceToHost));

 

	for (int i = 0; i < PatternCount; i++) {
		
		for (int j = 0; j < PatternLen; j++) {

			phi[i][j] = phi_1d[i*PatternLen + j];

		}

	}

	

	for (int i = 0; i < PatternCount; i++) {
		make_phi_inv(phi[i], phi_inv[i], PatternLen);

		make_E(p[i], phi_inv[i], E[i], PatternLen);
	}

	hipFree(dev_pattern_1d);

	hipFree(dev_phi_1d);

	hipFree(dev_temp_pattern_1d);

	delete[] phi_1d;

	delete[] temp_pattern_1d;


}

int finger_printing(int* p, int s, int m, int q) {

 

	int ret = 0;

 

	//for (int i = 0; i < q; i++) {

 

	int count;

	for (int j = s; j < s + q; j++) {

		count = 0;

		for (int k = s; k < j; k++) {

			if (p[k] <= p[j])

				count++;

		}

		ret += count * factorial(j - s);

	}

	//}

 

	return ret;

}

__device__ bool Check_OP(int* T, int arr_idx, int* P, int s, int len, int* phi_inv, int* E) {

 

	bool ret = true;

	for (int i = arr_idx; i < arr_idx + len - 1; i++) {

 

		if (E[i] == 0) {

			if (T[s + phi_inv[i]] >= T[s + phi_inv[i + 1]]) {

				ret = false;

				break;

			}

		}

		else {

			if (T[s + phi_inv[i]] != T[s + phi_inv[i + 1]]) {

				ret = false;

				break;

			}

		}

	}

	return ret;

}

__global__ void Search (int* match_count, bool * match, int* Text, int* p, int* Hash_Arr, int* phi_inv, int* E, int PatternCount, int PatternLen, int BlockSize, int TextLen) {

	int m = PatternLen;

	int q = BlockSize;


	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int totalthreadsize = blockDim.x * gridDim.x;

	int threadPerTextlen = (TextLen+totalthreadsize-1) / totalthreadsize;

	int start_idx = idx * threadPerTextlen; //Text start idx

	int end_idx = (idx + 1) *threadPerTextlen;//Text end idx 둘다 

	int s = start_idx-(m-q);

	 

	while (start_idx < end_idx) {

		if (start_idx < m - q) {

			start_idx++;

			continue;

		}

		if (start_idx >= TextLen - q) {

			break;

		}

		int temp = q_gram(Text, start_idx, m, q);

		for (int i = 0; i < PatternCount; i++) {

			if (temp == Hash_Arr[i]) {

				int P_len = find_len(p,i*m, PatternLen);

				if (Check_OP(Text,i*m, p, s, P_len, phi_inv, E)) {

					//match[TextLen*i + start_idx + q]=1;

					atomicAdd(&match_count[0], 2);
					match[start_idx+q] = true;

				}

			}

		}

		start_idx++;

		s++;

	}

	__syncthreads();

}
void InputData(int ** Pattern, int * Text, int PatternCount, int PatternLen, int TextLen, int FolderNumber){
	//Pattern input
	string pattern_filename = InputFolder + to_string(FolderNumber)+"/"+PatternInput + "_" + to_string(PatternCount) + "_" +to_string(PatternLen) + ".txt";
	ifstream pattern(pattern_filename);
	
	for (int i = 0; i < PatternCount; i++) {
    	for (int j = 0; j < PatternLen; j++) {
        	pattern >> Pattern[i][j];
		}
	}
	pattern.close();

	//Text input
	string text_filename = InputFolder + to_string(FolderNumber)+"/"+ TextInput + "_" + to_string(TextLen) + ".txt";
	ifstream text(text_filename);

	for (int i = 0; i < TextLen; i++) {
		text >> Text[i];
	}
	text.close();
	return ;
}

void PrintTestInfo(int PatternCount,int PatternLen,int TextLen, int MatchRes){
	printf("Pattern count: %d Pattern_length : %d TEXT SIZE : %d\nOP size : %d\n\n", PatternCount, PatternLen,TextLen, MatchRes);
}

int main() {

	int** Pattern;
	int* pattern_1d;
	int* PatternLength;
	int** phi;
	int** phi_inv;
	int* phi_inv_1d;
	int** E;
	int* E_1d;
	int* Text;
	int* hash_Arr;
	bool* match;
	int* match_count;

	struct inv_H * inverse_hash_Arr;

	// Calculated Table Size - 미리 계산된 q!
for(int FolderNumber = 0; FolderNumber <=2;FolderNumber++){
	for (int BlockSize = 3; BlockSize <= 3; BlockSize++) {
		for (int PatternCount = 100; PatternCount <= 1'000; PatternCount += 100) {
			for (int PatternLen = 3; PatternLen <= 15; PatternLen += 1) {
				for (int TextLen = 50'000; TextLen <= 50'000; TextLen += 10'000) {
					TotalStart = clock();

					Text = new int[TextLen];
					
					Pattern = new int*[PatternCount];
					for (int i = 0; i < PatternCount; i++) {
						Pattern[i] = new int[PatternLen];
					}
					//Read Text and Pattern
					InputData(Pattern, Text, PatternCount, PatternLen, TextLen,FolderNumber);
					hash_Arr = new int[PatternCount];
					inverse_hash_Arr = new inv_H[PatternCount];
					phi = new int *[PatternCount];
					phi_inv = new int *[PatternCount];
					E = new int *[PatternCount];
					PatternLength = new int[PatternCount];

					for (int i = 0; i < PatternCount; i++) {
						PatternLength[i] = PatternLen;
					}
					int res = 0;

 					for (int i = 0; i < PatternCount; i++)
					{
						res += PatternLength[i];
						phi[i] = new int[PatternLen];
						phi_inv[i] = new int[PatternLen];
						E[i] = new int[PatternLen];
					}
					pattern_1d = new int[res];

 
					int temp = 0;
					for (int i = 0; i < PatternCount; i++) {
						for (int j = 0; j < PatternLength[i]; j++) {
							pattern_1d[temp++] = Pattern[i][j];
						}
					}

					/* GPU 변수들 */

 					int* dev_text;
					int* dev_p;
					int* dev_hash_Arr;
					int* dev_phi_inv;
					int* dev_E;
					bool* dev_match;
					int* dev_match_count; 

					//********************************** finger 값 계산 *******************************************//

 					HANDLE_ERROR(hipMalloc((void**)&dev_p, res * sizeof(int)));//pattern
					HANDLE_ERROR(hipMalloc((void**)&dev_text, TextLen * sizeof(int)));
					//HANDLE_ERROR(hipMalloc((void**)&dev_p_length, PatternCount * sizeof(int)));
					HANDLE_ERROR(hipMalloc((void**)&dev_hash_Arr, PatternCount * sizeof(int)));
					HANDLE_ERROR(hipMalloc((void**)&dev_phi_inv, res * sizeof(int)));//make 1d arr!
					HANDLE_ERROR(hipMalloc((void**)&dev_E, res * sizeof(int)));
					HANDLE_ERROR(hipMalloc((void**)&dev_match, CopySize * sizeof(bool)));
					HANDLE_ERROR(hipMalloc((void**)&dev_match_count, 1 * sizeof(int)));

					HANDLE_ERROR(hipMemcpy(dev_p, pattern_1d, res * sizeof(int), hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(dev_text, Text, TextLen * sizeof(int), hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemset(dev_match_count, 0, 1 * sizeof(int)));
					HANDLE_ERROR(hipMemset(dev_match, 0, CopySize * sizeof(bool)));
					// PatternSet을 전처리하여 순위동형을 확인하는데 사용되는 phi_inverse, E 계산

					preprocessing_phi(pattern_1d, Pattern, phi, phi_inv, E, PatternCount, PatternLen);
 					// 각 패턴의 마지막 q그램을 계산하여 FingerPrint Table 생성

					preprocessing_table(Pattern, BlockSize, PatternCount, PatternLen, hash_Arr, inverse_hash_Arr);
 
					phi_inv_1d = new int[res];
					E_1d = new int[res];
					temp = 0;

					for (int i = 0; i < PatternCount; i++) {
						for (int j = 0; j < PatternLength[i]; j++) {
							phi_inv_1d[temp++] = phi_inv[i][j];
						}
					}

 					temp = 0;
					for (int i = 0; i < PatternCount; i++) {
						for (int j = 0; j < PatternLength[i]; j++) {
							E_1d[temp++] = E[i][j];
						}
					}

					HANDLE_ERROR(hipMemcpy(dev_phi_inv, phi_inv_1d, res * sizeof(int), hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(dev_E, E_1d, res * sizeof(int), hipMemcpyHostToDevice));
					HANDLE_ERROR(hipMemcpy(dev_hash_Arr, hash_Arr, PatternCount * sizeof(int), hipMemcpyHostToDevice));		
 
					SearchStart = clock();
					// 생성된 테이블로 Search 진행
					Search << < ((TextLen + 1023) / 1024), 1024 >> > (dev_match_count, dev_match, dev_text, dev_p, dev_hash_Arr, dev_phi_inv, dev_E, PatternCount, PatternLen, BlockSize, TextLen);
					SearchEnd = clock();
					hipDeviceSynchronize(); 

					//매치된 결과를 host에 복사

					match = new bool[CopySize];
					match_count = new int[1];
					CopyToHostStart = clock();
					HANDLE_ERROR(hipMemcpy(match_count, dev_match_count, 1 * sizeof(int), hipMemcpyDeviceToHost));
					HANDLE_ERROR(hipMemcpy(match, dev_match, CopySize * sizeof(bool), hipMemcpyDeviceToHost));
					CopyToHostEnd= clock();
					
					hipFree(dev_p);
					hipFree(dev_E);
					hipFree(dev_hash_Arr);
					hipFree(dev_phi_inv);
					hipFree(dev_text);
					hipFree(dev_match_count);
					hipFree(dev_match);

 					delete[] match;
					delete[] match_count;
					delete[] PatternLength;
					delete[] pattern_1d;
					delete[] E_1d;
					delete[] phi_inv_1d;
					delete[] hash_Arr;
					delete[] inverse_hash_Arr;

 					for (int i = 0; i < PatternCount; i++) {

						delete[] phi[i];

						delete[] phi_inv[i];

						delete[] E[i];

					}
					delete[] phi;
					delete[] phi_inv;
					delete[] E;
					for (int i = 0; i < PatternCount; i++) {

						delete[] Pattern[i];

					}
					delete[] Text;
					delete[] Pattern;

					TotalEnd = clock();
					PrintTestInfo(PatternCount, PatternLen,TextLen, match_count[0]);
					printf("Search Time : %fms\n",(double)(SearchEnd-SearchStart)/CLOCKS_PER_SEC);
					printf("Copy Time : %fms\n",(double)(CopyToHostEnd-CopyToHostStart)/CLOCKS_PER_SEC);
					printf("Total Time : %fms\n",(double)(TotalEnd-TotalStart)/CLOCKS_PER_SEC);
				}
			}

		}
	}
	}

	cout << endl;

	return 0;

}

 