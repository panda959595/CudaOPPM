#include "hip/hip_runtime.h"
/*

2020.01.02

검색단계 병렬화 correctness 확인완료

*/

#define _CRT_SECURE_NO_WARNINGS

#include<iostream>

#include<ctime>

#include<fstream>

#include<cstdlib>

#include<iomanip>

#include<cmath>

#include<string>

#include "cuda_by_example/common/book.h"

#include "hip/hip_runtime.h"

#include ""

 

using namespace std;

 

#define MAX_COUNT 2000

#define MAX_LEN 20

//#define BLOCK_SIZE 3

//#define TEXT_SIZE 1048575

 

#define min(a,b) a<b?a:b

#define max(a,b) a<b?b:a

 

class Hash_T {

public:

	int* pattern;

	Hash_T* next;

	int p_num;

	Hash_T();

};

Hash_T::Hash_T() {

 

	this->pattern = NULL;

	this->next = NULL;

	this->p_num = -1;

}

struct inv_H {

	int pattern_num;

	int FP_value;

};

 

int** make_p_prime(int** p, int m, int PATTERN_COUNT) {

 

	int** temp = new int*[PATTERN_COUNT];

	for (int i = 0; i < PATTERN_COUNT; i++) {

		temp[i] = new int[m];

	}

 

	for (int i = 0; i < PATTERN_COUNT; i++) {

		for (int j = 0; j < m; j++) {

			temp[i][j] = p[i][j];

		}

	}

 

	return temp;

}

void merge_H(int first, int mid, int last, inv_H* arr) {

 

	int idx = first;

 

	inv_H temp_arr[MAX_COUNT];

	int i = first, j = mid + 1;

 

	while (i <= mid && j <= last) {

		if (arr[i].FP_value <= arr[j].FP_value) {

			temp_arr[idx] = arr[i];

			idx++;

			i++;

		}

		else if (arr[i].FP_value > arr[j].FP_value) {

			temp_arr[idx] = arr[j];

			idx++;

			j++;

		}

	}

 

	if (i > mid) {

		for (int m = j; m <= last; m++) {

			temp_arr[idx] = arr[m];

			idx++;

		}

	}

	else {

		for (int m = i; m <= mid; m++) {

			temp_arr[idx] = arr[m];

			idx++;

		}

	}

 

	for (int m = first; m <= last; m++) {

		arr[m] = temp_arr[m];

	}

}

void mergeSort_H(int first, int last, inv_H* arr) {

	if (first < last) {

		int mid = (first + last) / 2;

		mergeSort_H(first, mid, arr);

		mergeSort_H(mid + 1, last, arr);

		merge_H(first, mid, last, arr);

	}

}

int Binary_Search(inv_H* arr, int size, int find_FP) {

 

	int low = 0, high = size - 1, mid;

 

	while (low <= high) {

		mid = (low + high) / 2;

		if (arr[mid].FP_value > find_FP) {

			high = mid - 1;

		}

		else if (arr[mid].FP_value < find_FP) {

			low = mid + 1;

		}

		else {

			return mid;

		}

	}

 

	return -1;

}

int factorial(int n) {

	return (n == 1 || n == 0) ? 1 : factorial(n - 1)*n;

}

 

__device__ int fac(int n) {

	return (n == 1 || n == 0) ? 1 : fac(n - 1)*n;

}

int q_gram_H(int* p, int s_idx, int m, int q) {

 

	//int size = m - q + 1;

	//int* ret = new int[size];

	int ret;

	int result;

	int count;

 

	//for (int i = 0; i < size; i++) {

 

	int i = s_idx;

	result = 0;

 

	for (int j = i; j < i + q; j++) {

		count = 0;

		for (int k = i; k < j; k++) {

			if (p[k] <= p[j])

				count++;

		}

		result += count * factorial(j - i);

	}

 

	ret = result;

	//}

	return ret;

}

 

__device__ int q_gram(int* p, int s_idx, int m, int q) {

 

	//int size = m - q + 1;

	//int* ret = new int[size];

	int ret;

	int result;

	int count;

 

	//for (int i = 0; i < size; i++) {

 

	int i = s_idx;

	result = 0;

 

	for (int j = i; j < i + q; j++) {

		count = 0;

		for (int k = i; k < j; k++) {

			if (p[k] <= p[j])

				count++;

		}

		result += count * fac(j - i);

	}

 

	ret = result;

	//}

	return ret;

}

void preprocessing_table(int** p, int B_size, int PATTERN_COUNT, int PATTERN_LEN, int* Hash_Arr, inv_H* inverse_Hash) {

 

	int m = PATTERN_LEN;

	int range = m - B_size + 1;

	int** p_prime = make_p_prime(p, m, PATTERN_COUNT);

 

	for (int i = 0; i < PATTERN_COUNT; i++) {

		Hash_Arr[i] = q_gram_H(p_prime[i], range - 1, m, B_size);

	}

	for (int i = 0; i < PATTERN_COUNT; i++)

		delete[] p_prime[i];

	delete[] p_prime;

}

 

int find_len_H(int* p, int PATTERN_LEN) {

 

	int ret = PATTERN_LEN;

	for (int i = 0; i < PATTERN_LEN; i++) {

		if (p[i] < 0 || p[i] == 0) {

			ret = i;

			break;

		}

	}

 

	return ret;

}

 

__device__ int find_len(int* p, int arr_idx, int PATTERN_LEN) {

 

	int ret = PATTERN_LEN;

	for (int i = arr_idx; i < PATTERN_LEN + arr_idx; i++) {

		if (p[i] < 0 || p[i] == 0) {

			ret = i;

			break;

		}

	}

	return ret;

}

__device__ int find_max_H(int* p,int s_idx, int len) {

 

	int ret = 0;

 

	for (int i = s_idx; i < s_idx+len; i++) {

		if (p[i] > ret)

			ret = p[i];

	}

 

	return ret;

}

int find_max(int* p, int len) {

 

	int ret = 0;

 

	for (int i = 0; i < len; i++) {

		if (p[i] > ret)

			ret = p[i];

	}

 

	return ret;

}

void merge(int first, int mid, int last, int* arr) {

 

	int idx = first;

	int temp_arr[MAX_COUNT];

 

 

	int i = first, j = mid + 1;

 

	while (i <= mid && j <= last) {

		if (arr[i] <= arr[j]) {

			temp_arr[idx] = arr[i];

			idx++;

			i++;

		}

		else if (arr[i] > arr[j]) {

			temp_arr[idx] = arr[j];

			idx++;

			j++;

		}

	}

 

	if (i > mid) {

		for (int m = j; m <= last; m++) {

			temp_arr[idx] = arr[m];

			idx++;

		}

	}

	else {

		for (int m = i; m <= mid; m++) {

			temp_arr[idx] = arr[m];

			idx++;

		}

	}

 

	for (int m = first; m <= last; m++) {

		arr[m] = temp_arr[m];

	}

}

void mergeSort(int first, int last, int* arr) {

 

	if (first < last) {

		int mid = (first + last) / 2;

		mergeSort(first, mid, arr);

		mergeSort(mid + 1, last, arr);

		merge(first, mid, last, arr);

	}

}

 

void make_phi_inv(int* phi, int* phi_inv, int len) {

 

	for (int i = 0; i < len; i++) {

		phi_inv[phi[i]] = i;

	}

}

void make_E(int* p, int* phi_inv, int* E, int len) {

 

	for (int i = 0; i < len - 1; i++) {

		if (p[phi_inv[i]] == p[phi_inv[i + 1]])

			E[i] = 1;

		else

			E[i] = 0;

	}

}

__global__ void make_phi(int* temp_p_1d, int* p_1d, int* phi_1d, int len, int PATTERN_COUNT) {

	//temp_p_1d 가 정렬되어있는것임

	//하나의 스레드가 하나의 파이 만든다

	int idx = blockDim.x*blockIdx.x + threadIdx.x;

	int arr_idx = idx * len;

 

	if (idx <PATTERN_COUNT) {

		int * flag = new int[len];

		for (int i = 0; i < len; i++) {

			flag[i] = 0;

		}

		for (int i = arr_idx; i < arr_idx + len; i++) {

			phi_1d[i] = -1;

		}

 

		int temp;

		for (int i = arr_idx; i < arr_idx+len; i++) {

			temp = p_1d[i];

			for (int j = arr_idx; j < arr_idx+len; j++) {

				if (temp_p_1d[j] == temp && flag[j-arr_idx] == 0) {// temp_p_1d는 정렬되어 있음 

					phi_1d[i] = j - arr_idx;

					flag[j-arr_idx] = 1;

					break;

				}

			}

		}

		delete flag;

	}

}

 

void preprocessing_phi(int* pattern_1d, int** p, int** phi, int** phi_inv, int** E, int PATTERN_COUNT, int PATTERN_LEN) {

	//string FOLDER = "./OUTPUT/";

	//string FILENAME = "out.txt";

	int* temp_pattern_1d = new int[PATTERN_COUNT * PATTERN_LEN];

	int* temp_arr = new int[PATTERN_LEN];

	int* phi_1d = new int[PATTERN_COUNT *PATTERN_LEN];

 

	//global 함수//

	int* dev_pattern_1d;

	int* dev_temp_pattern_1d;

	int* dev_phi_1d;

	//gpu 메모리 할당//

	HANDLE_ERROR(hipMalloc((void**)&dev_pattern_1d, PATTERN_COUNT * PATTERN_LEN * sizeof(int)));

	HANDLE_ERROR(hipMalloc((void**)&dev_temp_pattern_1d, PATTERN_COUNT * PATTERN_LEN * sizeof(int)));

	HANDLE_ERROR(hipMalloc((void**)&dev_phi_1d, PATTERN_COUNT * PATTERN_LEN * sizeof(int)));

 

	for (int i = 0; i < PATTERN_COUNT; i++) {

		for (int j = 0; j < PATTERN_LEN; j++) {

			temp_arr[j] = pattern_1d[i*PATTERN_LEN+j];

		}

		mergeSort(0, PATTERN_LEN - 1, temp_arr);

 

		for (int j = 0; j < PATTERN_LEN; j++) {

			temp_pattern_1d[i*PATTERN_LEN + j] = temp_arr[j];

		}

	}

 

	HANDLE_ERROR(hipMemcpy(dev_pattern_1d, pattern_1d, PATTERN_COUNT * PATTERN_LEN * sizeof(int), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(dev_temp_pattern_1d, temp_pattern_1d, PATTERN_COUNT * PATTERN_LEN * sizeof(int),hipMemcpyHostToDevice));

 

	make_phi << <(PATTERN_COUNT + 127) / 128, 128 >> > (dev_temp_pattern_1d, dev_pattern_1d, dev_phi_1d, PATTERN_LEN, PATTERN_COUNT);

	hipDeviceSynchronize();

	HANDLE_ERROR(hipMemcpy(phi_1d, dev_phi_1d, PATTERN_COUNT * PATTERN_LEN * sizeof(int), hipMemcpyDeviceToHost));

 

	for (int i = 0; i < PATTERN_COUNT; i++) {
		make_phi
		for (int j = 0; j < PATTERN_LEN; j++) {

			phi[i][j] = phi_1d[i*PATTERN_LEN + j];

		}

	}

	

	for (int i = 0; i < PATTERN_COUNT; i++) {

 

		make_phi_inv(phi[i], phi_inv[i], PATTERN_LEN);

		make_E(p[i], phi_inv[i], E[i], PATTERN_LEN);

 

	}

	hipFree(dev_pattern_1d);

	hipFree(dev_phi_1d);

	hipFree(dev_temp_pattern_1d);

	delete[] phi_1d;

	delete[] temp_pattern_1d;

	cout << "preprocessing 1d : " << ee - ss << endl;

	cout << "preprocessing phi_inv, E : " << e - s << endl;

 

	/*string output = FOLDER + FILENAME;

	ofstream out(output);

	for (int i = 0; i < PATTERN_COUNT; i++) {

		for (int j = 0; j < PATTERN_LEN; j++) {

			out << phi_1d[i*PATTERN_LEN + j] << " ";

		}

		out << "\n";

	}

	out.close();*/

 

}

int finger_printing(int* p, int s, int m, int q) {

 

	int ret = 0;

 

	//for (int i = 0; i < q; i++) {

 

	int count;

	for (int j = s; j < s + q; j++) {

		count = 0;

		for (int k = s; k < j; k++) {

			if (p[k] <= p[j])

				count++;

		}

		ret += count * factorial(j - s);

	}

	//}

 

	return ret;

}

__device__ bool Check_OP(int* T, int arr_idx, int* P, int s, int len, int* phi_inv, int* E) {

 

	bool ret = true;

	for (int i = arr_idx; i < arr_idx + len - 1; i++) {

 

		if (E[i] == 0) {

			if (T[s + phi_inv[i]] >= T[s + phi_inv[i + 1]]) {

				ret = false;

				break;

			}

		}

		else {

			if (T[s + phi_inv[i]] != T[s + phi_inv[i + 1]]) {

				ret = false;

				break;

			}

		}

	}

	return ret;

}

__global__ void Search (int* match_count, int* match, int* Text, int* p, int* Hash_Arr, int* phi_inv, int* E, int PATTERN_COUNT, int PATTERN_LEN, int BLOCK_SIZE, int TEXT_SIZE) {

	int m = PATTERN_LEN;

	int q = BLOCK_SIZE;

 

	int bidx = blockIdx.x;

	int tidx = threadIdx.x;

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int totalthreadsize = blockDim.x * gridDim.x;

	int threadPerTextlen = (TEXT_SIZE+totalthreadsize-1) / totalthreadsize;

	int start_idx = idx * threadPerTextlen; //Text start idx

	int end_idx = (idx + 1) *threadPerTextlen;//Text end idx 둘다 

	int s = start_idx-(m-q);

	 

	while (start_idx < end_idx) {

		if (start_idx < m - q) {

			start_idx++;

			continue;

		}

		if (start_idx >= TEXT_SIZE - q) {

			break;

		}

		 

		int temp = q_gram(Text, start_idx, m, q);

		for (int i = 0; i < PATTERN_COUNT; i++) {

			if (temp == Hash_Arr[i]) {

				int P_len = find_len(p,i*m, PATTERN_LEN);

				if (Check_OP(Text,i*m, p, s, P_len, phi_inv, E)) {

					//match[TEXT_SIZE*i + start_idx + q]=1;

					atomicAdd(&match_count[0], 2);

					atomicExch(&(match[match_count[0] - 2]), i);

					atomicExch(&(match[match_count[0] - 1]), start_idx + q);

				}

			}

		}

		start_idx++;

		s++;

	}

	__syncthreads();

}

 

int main() {

 

	clock_t makephi_stime, makephi_etime;

	clock_t maketable_stime, maketable_etime;

	clock_t search_stime, search_etime;

	clock_t total_stime, total_etime;

 

	int** PATTERN_SET;

	int* pattern_1d;

	int* pattern_length;

	int** phi;

	int** phi_inv;

	int* phi_inv_1d;

	int** E;

	int* E_1d;

	int* Text;

	int* hash_Arr;

	int* match;

	int* match_count;

	struct inv_H * inverse_hash_Arr;

 

	// Calculated Table Size - 미리 계산된 q!

	int TABLE_SIZE[10] = { 0, 0, 0, 6, 24, 120, 720, 5040, 40320, 362880 }; // Q : 3 ~ 9

 

	// Set Files Name and Folder Name

	string TC_FOLDER = "./TESTCASE/";

	string TEXT_FILE = "TextSample";

	string PATTERN_FILE = "IntStr";

	string TIME_FOLDER = "./TIME/";

	string TIME_FILE = "TimeRecord_";

	string check = "check";

 

	// PATTERN_COUNT : 패턴 개수 ( k )

	// PATTERN_LEN : 패턴 길이 ( m )

 

	string time_filename = TIME_FOLDER + check + ".txt";

	ofstream out(time_filename);

 

	for (int BLOCK_SIZE = 5; BLOCK_SIZE <= 5; BLOCK_SIZE++) {

		for (int PATTERN_COUNT = 1'000; PATTERN_COUNT <= 5'000; PATTERN_COUNT += 1'000) {

			for (int PATTERN_LEN = 6; PATTERN_LEN <= 10; PATTERN_LEN += 1) {

				for (int TEXT_SIZE = 10'000; TEXT_SIZE <= 50'000; TEXT_SIZE += 10'000) {

					double phi_time = 0;

					double search_time = 0;

					double maketable_time = 0;

					double total_time = 0;

 

					// Read Pattern Information - 패턴개수와 패턴길이에 맞게 패턴 파일 읽음

					string pattern_filename = TC_FOLDER + PATTERN_FILE + "_" + to_string(PATTERN_COUNT) + "_" + to_string(PATTERN_LEN) + ".txt";

					ifstream pattern(pattern_filename);

 

					PATTERN_SET = new int*[PATTERN_COUNT

					];

					for (int i = 0; i < PATTERN_COUNT; i++) {

						PATTERN_SET[i] = new int[PATTERN_LEN];

					}

					for (int i = 0; i < PATTERN_COUNT; i++) {

						for (int j = 0; j < PATTERN_LEN; j++) {

							pattern >> PATTERN_SET[i][j];

						}

					}

					pattern.close();

 

					// Read Text Information - 텍스트 파일 읽음

					string text_filename = TC_FOLDER + TEXT_FILE + "_" + to_string(TEXT_SIZE) + ".txt";

					ifstream text(text_filename);

					Text = new int[TEXT_SIZE];

					for (int i = 0; i < TEXT_SIZE; i++) {

						text >> Text[i];

					}

					text.close();

					/****************************************/

 

					// 전처리 단계에서 사용될 Array 초기화

 

					total_stime = clock();

 

					hash_Arr = new int[PATTERN_COUNT];

					inverse_hash_Arr = new inv_H[PATTERN_COUNT];

					phi = new int *[PATTERN_COUNT];

					phi_inv = new int *[PATTERN_COUNT];

					E = new int *[PATTERN_COUNT];

 

					pattern_length = new int[PATTERN_COUNT];

					for (int i = 0; i < PATTERN_COUNT; i++) {

						pattern_length[i] = PATTERN_LEN;

					}

 

					int res = 0;

 

					for (int i = 0; i < PATTERN_COUNT; i++)

					{

						res += pattern_length[i];

						phi[i] = new int[PATTERN_LEN];

						phi_inv[i] = new int[PATTERN_LEN];

						E[i] = new int[PATTERN_LEN];

					}

					pattern_1d = new int[res];

 

					int temp = 0;

					for (int i = 0; i < PATTERN_COUNT; i++) {

						for (int j = 0; j < pattern_length[i]; j++) {

							pattern_1d[temp++] = PATTERN_SET[i][j];

						}

					}

 

					/* GPU 변수들 */

 

					int* dev_text;

					int* dev_p;

					int* dev_hash_Arr;

					int* dev_phi_inv;

					int* dev_E;

					int* dev_match;

					int* dev_match_count; 

 

					//********************************** finger 값 계산 *******************************************//

 

					HANDLE_ERROR(hipMalloc((void**)&dev_p, res * sizeof(int)));//pattern

					HANDLE_ERROR(hipMalloc((void**)&dev_text, TEXT_SIZE * sizeof(int)));

					//HANDLE_ERROR(hipMalloc((void**)&dev_p_length, PATTERN_COUNT * sizeof(int)));

					HANDLE_ERROR(hipMalloc((void**)&dev_hash_Arr, PATTERN_COUNT * sizeof(int)));

					HANDLE_ERROR(hipMalloc((void**)&dev_phi_inv, res * sizeof(int)));//make 1d arr!

					HANDLE_ERROR(hipMalloc((void**)&dev_E, res * sizeof(int)));

					HANDLE_ERROR(hipMalloc((void**)&dev_match, 5 * 1'000'000 * sizeof(int)));

					HANDLE_ERROR(hipMalloc((void**)&dev_match_count, 1 * sizeof(int)));

 

					//copy_stime = clock();

					HANDLE_ERROR(hipMemcpy(dev_p, pattern_1d, res * sizeof(int), hipMemcpyHostToDevice));

					HANDLE_ERROR(hipMemcpy(dev_text, Text, TEXT_SIZE * sizeof(int), hipMemcpyHostToDevice));

					HANDLE_ERROR(hipMemset(dev_match_count, 0, 1 * sizeof(int)));

					HANDLE_ERROR(hipMemset(dev_match, 0, 5*1'000'000 * sizeof(int)));

					// PatternSet을 전처리하여 순위동형을 확인하는데 사용되는 phi_inverse, E 계산

					makephi_stime = clock();

					preprocessing_phi(pattern_1d, PATTERN_SET, phi, phi_inv, E, PATTERN_COUNT, PATTERN_LEN);

					makephi_etime = clock();

 

					// 각 패턴의 마지막 q그램을 계산하여 FingerPrint Table 생성

					maketable_stime = clock();

					preprocessing_table(PATTERN_SET, BLOCK_SIZE, PATTERN_COUNT, PATTERN_LEN, hash_Arr, inverse_hash_Arr);

					maketable_etime = clock();

 

					phi_inv_1d = new int[res];

					E_1d = new int[res];

					temp = 0;

					for (int i = 0; i < PATTERN_COUNT; i++) {

						for (int j = 0; j < pattern_length[i]; j++) {

							phi_inv_1d[temp++] = phi_inv[i][j];

						}

					}

 

					temp = 0;

					for (int i = 0; i < PATTERN_COUNT; i++) {

						for (int j = 0; j < pattern_length[i]; j++) {

							E_1d[temp++] = E[i][j];

						}

					}

					HANDLE_ERROR(hipMemcpy(dev_phi_inv, phi_inv_1d, res * sizeof(int), hipMemcpyHostToDevice));

					HANDLE_ERROR(hipMemcpy(dev_E, E_1d, res * sizeof(int), hipMemcpyHostToDevice));

					HANDLE_ERROR(hipMemcpy(dev_hash_Arr, hash_Arr, PATTERN_COUNT * sizeof(int), hipMemcpyHostToDevice));

 

					

 

					// 생성된 테이블로 Search 진행

					search_stime = clock();

					Search << < ((TEXT_SIZE + 1023) / 1024), 1024 >> > (dev_match_count, dev_match, dev_text, dev_p, dev_hash_Arr, dev_phi_inv, dev_E, PATTERN_COUNT, PATTERN_LEN, BLOCK_SIZE, TEXT_SIZE);

					hipDeviceSynchronize();

					search_etime = clock();

 

					//매치된 결과를 host에 복사

					match = new int[5 * 1'000'000];

					match_count = new int[1];

					HANDLE_ERROR(hipMemcpy(match_count, dev_match_count, 1 * sizeof(int), hipMemcpyDeviceToHost));

					HANDLE_ERROR(hipMemcpy(match, dev_match, 5 * 1'000'000 * sizeof(int), hipMemcpyDeviceToHost));

					

					int host_match_count = match_count[0];

					out << match_count[0] << "\n";

 

					/*for (int col = 0; col < PATTERN_COUNT; col++) {

							for (int row = 0; row < PATTERN_LEN; row++) {

								out <<phi[col][row] << " ";

							}

							out << "\n";

					}*/

 

					total_etime = clock();

 

					hipFree(dev_p);

					hipFree(dev_E);

					hipFree(dev_hash_Arr);

					hipFree(dev_phi_inv);

					hipFree(dev_text);

					hipFree(dev_match_count);

					hipFree(dev_match);

 

					delete[] match;

					delete[] match_count;

					delete[] pattern_length;

					delete[] pattern_1d;

					delete[] E_1d;

					delete[] phi_inv_1d;

					delete[] hash_Arr;

					delete[] inverse_hash_Arr;

 

					for (int i = 0; i < PATTERN_COUNT; i++) {

						delete[] phi[i];

						delete[] phi_inv[i];

						delete[] E[i];

					}

					delete[] phi;

					delete[] phi_inv;

					delete[] E;

 

					for (int i = 0; i < PATTERN_COUNT; i++) {

						delete[] PATTERN_SET[i];

					}

					delete[] Text;

					delete[] PATTERN_SET;

 

					printf("Pattern count: %d Pattern_length : %d\n TEXT SIZE : %d\n", PATTERN_COUNT, PATTERN_LEN, TEXT_SIZE);

					printf("Make PI Time : %3.10f ms\n", (float)makephi_etime - makephi_stime);

					printf("Search Time : %3.10f ms\n", (float)search_etime - search_stime);

					printf("Table Time : %3.10f ms\n", (float)maketable_etime - maketable_stime);

					printf("Total TIme : %3.10f ms\n\n", (float)total_etime - total_stime);

				}

			}

		}

	}

	cout << endl;

	return 0;

}

 