#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include "cuda_by_example/common/book.h"
#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include<sys/time.h>
#include<fstream>
#include<cstdlib>
#include<ctime>
#include<iomanip>
#include<cmath>
#include<string>

using namespace std;
int a = 0;
#define MAX_COUNT 2000
#define MAX_LEN 20
#define Repeat 10
//#define BLOCK_SIZE 3
//#define TEXT_SIZE 1048575

#define min(a,b) a<b?a:b
#define max(a,b) a<b?b:a

string InputFolder = "./TESTCASE/TC-";
string OutputFolder = "./JournalV4OUTPUT/TC-";
string TimeFolder = "./JournalV4TIME/";
string TextInput = "TextSample";
string PatternInput = "IntStr";
string TimeInput = "TimeRecord_";

__constant__ int dev_phi_inv[16'000]; //MAX

void OutputData(int PatternCount, int PatternLen, int TextLen, int BlockSize, int FolderNumber, int MatchRes) {
	string FileName = OutputFolder + to_string(FolderNumber) + "/" + PatternInput + "_" +
		to_string(PatternCount) + "_" + to_string(PatternLen) + "_" + to_string(TextLen) + "_" + to_string(BlockSize) + ".txt";

	ofstream FileStream(FileName);
	FileStream << MatchRes;
	/*FileStream<<"\n";
	for(int t=0;t<TextLen; t++){
		FileStream<<MatchResDetail[t]<<" ";
	}*/
	FileStream.close();
}

void OutputTime(double Pre, float Search, double Total, double TotalCopy, int PatternCount, int PatternLen, int TextLen, int BlockSize) {
	string FileName = TimeFolder + PatternInput + "_" +
		to_string(PatternCount) + "_" + to_string(PatternLen) + "_" +
		to_string(TextLen) + "_" + to_string(BlockSize) + ".txt";

	ofstream FileStream(FileName);
	FileStream << (double)(Pre) / Repeat << " " << (double)(Search) / Repeat << " "
		<< (double)(Total) / Repeat << " " << (double)(TotalCopy) / Repeat;

	FileStream.close();
}


class Hash_T {
public:
	int* pattern;
	Hash_T* next;
	int p_num;
	Hash_T();
};
Hash_T::Hash_T() {

	this->pattern = NULL;
	this->next = NULL;
	this->p_num = -1;
}
struct inv_H {
	int pattern_num;
	int FP_value;
};

int** make_p_prime(int** p, int m, int PATTERN_COUNT) {

	int** temp = new int*[PATTERN_COUNT];
	for (int i = 0; i < PATTERN_COUNT; i++) {
		temp[i] = new int[m];
	}

	for (int i = 0; i < PATTERN_COUNT; i++) {
		for (int j = 0; j < m; j++) {
			temp[i][j] = p[i][j];
		}
	}

	return temp;
}
void merge_H(int first, int mid, int last, inv_H* arr) {

	int idx = first;

	inv_H temp_arr[MAX_COUNT];
	int i = first, j = mid + 1;

	while (i <= mid && j <= last) {
		if (arr[i].FP_value <= arr[j].FP_value) {
			temp_arr[idx] = arr[i];
			idx++;
			i++;
		}
		else if (arr[i].FP_value > arr[j].FP_value) {
			temp_arr[idx] = arr[j];
			idx++;
			j++;
		}
	}

	if (i > mid) {
		for (int m = j; m <= last; m++) {
			temp_arr[idx] = arr[m];
			idx++;
		}
	}
	else {
		for (int m = i; m <= mid; m++) {
			temp_arr[idx] = arr[m];
			idx++;
		}
	}

	for (int m = first; m <= last; m++) {
		arr[m] = temp_arr[m];
	}
}
void mergeSort_H(int first, int last, inv_H* arr) {
	if (first < last) {
		int mid = (first + last) / 2;
		mergeSort_H(first, mid, arr);
		mergeSort_H(mid + 1, last, arr);
		merge_H(first, mid, last, arr);
	}
}
int Binary_Search(inv_H* arr, int size, int find_FP) {

	int low = 0, high = size - 1, mid;

	while (low <= high) {
		mid = (low + high) / 2;
		if (arr[mid].FP_value > find_FP) {
			high = mid - 1;
		}
		else if (arr[mid].FP_value < find_FP) {
			low = mid + 1;
		}
		else {
			return mid;
		}
	}

	return -1;
}
int factorial(int n) {
	return (n == 1 || n == 0) ? 1 : factorial(n - 1)*n;
}

__device__ int fac(int n) {
	return (n == 1 || n == 0) ? 1 : fac(n - 1)*n;
}
int q_gram_H(int* p, int s_idx, int m, int q) {

	//int size = m - q + 1;
	//int* ret = new int[size];
	int ret;
	int result;
	int count;

	//for (int i = 0; i < size; i++) {

	int i = s_idx;
	result = 0;

	for (int j = i; j < i + q; j++) {
		count = 0;
		for (int k = i; k < j; k++) {
			if (p[k] <= p[j])
				count++;
		}
		result += count * factorial(j - i);
	}

	ret = result;
	//}
	return ret;
}

__device__ int q_gram(int* p, int s_idx, int m, int q) {

	//int size = m - q + 1;
	//int* ret = new int[size];
	int ret;
	int result;
	int count;

	//for (int i = 0; i < size; i++) {

	int i = s_idx;
	result = 0;

	for (int j = i; j < i + q; j++) {
		count = 0;
		for (int k = i; k < j; k++) {
			if (p[k] <= p[j])
				count++;
		}
		result += count * fac(j - i);
	}

	ret = result;
	//}
	return ret;
}
void preprocessing_table(int** p, int B_size, int PATTERN_COUNT, int PATTERN_LEN, int* Hash_Arr, inv_H* inverse_Hash) {

	int m = PATTERN_LEN;
	int range = m - B_size + 1;
	int** p_prime = make_p_prime(p, m, PATTERN_COUNT);

	for (int i = 0; i < PATTERN_COUNT; i++) {
		Hash_Arr[i] = q_gram_H(p_prime[i], range - 1, m, B_size);
	}
	for (int i = 0; i < PATTERN_COUNT; i++)
		delete[] p_prime[i];
	delete[] p_prime;
}

int find_len_H(int* p, int PATTERN_LEN) {

	int ret = PATTERN_LEN;
	for (int i = 0; i < PATTERN_LEN; i++) {
		if (p[i] < 0 || p[i] == 0) {
			ret = i;
			break;
		}
	}

	return ret;
}

__device__ int find_len(int* p, int arr_idx, int PATTERN_LEN) {

	int ret = PATTERN_LEN;
	for (int i = arr_idx; i < PATTERN_LEN + arr_idx; i++) {
		if (p[i] < 0 || p[i] == 0) {
			ret = i;
			break;
		}
	}
	return ret;
}

int find_max(int* p, int len) {

	int ret = 0;

	for (int i = 0; i < len; i++) {
		if (p[i] > ret)
			ret = p[i];
	}

	return ret;
}
void merge(int first, int mid, int last, int* arr) {

	int idx = first;
	int temp_arr[MAX_COUNT];


	int i = first, j = mid + 1;

	while (i <= mid && j <= last) {
		if (arr[i] <= arr[j]) {
			temp_arr[idx] = arr[i];
			idx++;
			i++;
		}
		else if (arr[i] > arr[j]) {
			temp_arr[idx] = arr[j];
			idx++;
			j++;
		}
	}

	if (i > mid) {
		for (int m = j; m <= last; m++) {
			temp_arr[idx] = arr[m];
			idx++;
		}
	}
	else {
		for (int m = i; m <= mid; m++) {
			temp_arr[idx] = arr[m];
			idx++;
		}
	}

	for (int m = first; m <= last; m++) {
		arr[m] = temp_arr[m];
	}
}
void mergeSort(int first, int last, int* arr) {

	if (first < last) {
		int mid = (first + last) / 2;
		mergeSort(first, mid, arr);
		mergeSort(mid + 1, last, arr);
		merge(first, mid, last, arr);
	}
}
void make_phi(int* temp_p, int* p, int* phi, int len) { 

	int max_val = find_max(p, len);
	int* flag = new int[max_val + 1];
	for (int i = 0; i <= max_val; i++) {
		flag[i] = 0;
	}

	int temp;

	for (int i = 0; i < len; i++) {
		temp = p[i];
		for (int j = flag[temp]; j < len; j++) {
			if (temp_p[j] == temp) {
				flag[temp] = j + 1;
				phi[i] = j;
				break;
			}
		}
	}

	delete[]flag;
}
void make_phi_inv(int* phi, int* phi_inv, int len) {

	for (int i = 0; i < len; i++) {
		phi_inv[phi[i]] = i;
	}
}
void make_E(int* p, int* phi_inv, int* E, int len) {

	for (int i = 0; i < len - 1; i++) {
		if (p[phi_inv[i]] == p[phi_inv[i + 1]])
			E[i] = 1;
		else
			E[i] = 0;
	}
}
void preprocessing_phi(int** p, int** phi, int** phi_inv, int** E, int PATTERN_COUNT, int PATTERN_LEN) {

	int len;
	int* temp_arr;
	for (int i = 0; i < PATTERN_COUNT; i++) {

		len = find_len_H(p[i], PATTERN_LEN);
		temp_arr = new int[len];

		for (int j = 0; j < len; j++) {
			temp_arr[j] = p[i][j];
		}
		mergeSort(0, len - 1, temp_arr);

		make_phi(temp_arr, p[i], phi[i], len);

		make_phi_inv(phi[i], phi_inv[i], len);

		make_E(p[i], phi_inv[i], E[i], len);

		delete[]temp_arr;
	}
}
int finger_printing(int* p, int s, int m, int q) {

	int ret = 0;

	//for (int i = 0; i < q; i++) {

	int count;
	for (int j = s; j < s + q; j++) {
		count = 0;
		for (int k = s; k < j; k++) {
			if (p[k] <= p[j])
				count++;
		}
		ret += count * factorial(j - s);
	}
	//}

	return ret;
}
__device__ bool Check_OP(int* T, int arr_idx, int* P, int s, int len, int* E) {

	bool ret = true;
	for (int i = arr_idx; i < arr_idx + len - 1; i++) {

		if (E[i] == 0) {
			if (T[s + dev_phi_inv[i]] >= T[s + dev_phi_inv[i + 1]]) {
				ret = false;
				break;
			}
		}
		else {
			if (T[s + dev_phi_inv[i]] != T[s + dev_phi_inv[i + 1]]) {
				ret = false;
				break;
			}
		}
	}
	return ret;
}
__global__ void Search (int* match_count, int* match, int* Text, int* p, int* Hash_Arr, int* E, int PATTERN_COUNT, int PATTERN_LEN, int BLOCK_SIZE, int TEXT_SIZE) {
	int m = PATTERN_LEN;
	int q = BLOCK_SIZE;

	int bidx = blockIdx.x;
	int tidx = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int totalthreadsize = blockDim.x * gridDim.x;
	int threadPerTextlen = (TEXT_SIZE+totalthreadsize-1) / totalthreadsize;
	int start_idx = idx * threadPerTextlen; //Text start idx
	int end_idx = (idx + 1) *threadPerTextlen;//Text end idx �Ѵ� 
	int s = start_idx-(m-q);
	while (start_idx < end_idx) {
		if (start_idx < m - q) {
			start_idx++;
			continue;
		}
		if (start_idx >= TEXT_SIZE - q) {
			break;
		}

		int temp = q_gram(Text, start_idx, m, q);
		for (int i = 0; i < PATTERN_COUNT; i++) {
			if (temp == Hash_Arr[i]) {
				int P_len = find_len(p,i*m, PATTERN_LEN);
				if (Check_OP(Text,i*m, p, s, P_len, E)) {
					//match[TEXT_SIZE*i + start_idx + q]=1;
					atomicAdd(&match_count[0], 1);
					/*atomicExch(&(match[match_count[0] - 2]), i);
					atomicExch(&(match[match_count[0] - 1]), start_idx + q);*/
				}
			}
		}
		start_idx++;
		s++;
	}
	__syncthreads();
}

int main() {

	struct timeval PreStart, PreEnd, SearchStart, SearchEnd, TotalStart, TotalEnd, CopyToHostStart, CopyToHostEnd;

	int** PATTERN_SET;
	int* pattern_1d;
	int* pattern_length;
	int** phi;
	int** phi_inv;
	int* phi_inv_1d;
	int** E;
	int* E_1d;
	int* Text;
	int* hash_Arr;
	int* match;
	int* match_count;
	struct inv_H * inverse_hash_Arr;

	// Calculated Table Size - �̸� ���� q!
	int TABLE_SIZE[10] = { 0, 0, 0, 6, 24, 120, 720, 5040, 40320, 362880 }; // Q : 3 ~ 9

	// Set Files Name and Folder Name
	string TC_FOLDER = "./TESTCASE/";
	string TEXT_FILE = "TextSample";
	string PATTERN_FILE = "IntStr";
	string TIME_FOLDER = "./TIME/";
	string TIME_FILE = "TimeRecord_";

	// PATTERN_COUNT : ���� ���� ( k )
	// PATTERN_LEN : ���� ���� ( m )

	for (int BLOCK_SIZE = 7; BLOCK_SIZE <= 7; BLOCK_SIZE++) {
		for (int PATTERN_COUNT = 100; PATTERN_COUNT <= 1'000; PATTERN_COUNT += 100) {
			for (int PATTERN_LEN = 7; PATTERN_LEN <= 15; PATTERN_LEN += 1) {
				printf("Pattern Count: %d\nPattern Len : %d\n",PATTERN_COUNT, PATTERN_LEN);
for (int TEXT_SIZE = 100'000; TEXT_SIZE <= 1'000'000; TEXT_SIZE += 100'000) {
					double sec, usec;
					double TotalPre = 0;
					double TotalSearch = 0;
					double Total = 0;
					double TotalCopy = 0;
					for (int FolderNumber = 0; FolderNumber < Repeat; FolderNumber++) {
						// Read Pattern Information - ���ϰ����� ���ϱ��̿� �°� ���� ���� ����
						string pattern_filename = InputFolder + to_string(FolderNumber) + "/" + PatternInput + "_" + to_string(PATTERN_COUNT) + "_" + to_string(PATTERN_LEN) + ".txt";

						ifstream pattern(pattern_filename);

						PATTERN_SET = new int* [PATTERN_COUNT];
						for (int i = 0; i < PATTERN_COUNT; i++) {
							PATTERN_SET[i] = new int[PATTERN_LEN];
						}
						for (int i = 0; i < PATTERN_COUNT; i++) {
							for (int j = 0; j < PATTERN_LEN; j++) {
								pattern >> PATTERN_SET[i][j];
							}
						}
						pattern.close();

						// Read Text Information - �ؽ�Ʈ ���� ����
						string text_filename = InputFolder + to_string(FolderNumber) + "/" + TextInput + "_" + to_string(TEXT_SIZE) + ".txt";
						ifstream text(text_filename);
						Text = new int[TEXT_SIZE];
						for (int i = 0; i < TEXT_SIZE; i++) {
							text >> Text[i];
						}
						text.close();
						/****************************************/

						// ��ó�� �ܰ迡�� ���� Array �ʱ�ȭ

						gettimeofday(&TotalStart, NULL);

						hash_Arr = new int[PATTERN_COUNT];
						inverse_hash_Arr = new inv_H[PATTERN_COUNT];
						phi = new int* [PATTERN_COUNT];
						phi_inv = new int* [PATTERN_COUNT];
						E = new int* [PATTERN_COUNT];

						pattern_length = new int[PATTERN_COUNT];
						for (int i = 0; i < PATTERN_COUNT; i++) {
							pattern_length[i] = PATTERN_LEN;
						}

						int res = 0;

						for (int i = 0; i < PATTERN_COUNT; i++)
						{
							res += pattern_length[i];
							phi[i] = new int[PATTERN_LEN];
							phi_inv[i] = new int[PATTERN_LEN];
							E[i] = new int[PATTERN_LEN];
						}
						pattern_1d = new int[res];

						int temp = 0;
						for (int i = 0; i < PATTERN_COUNT; i++) {
							for (int j = 0; j < pattern_length[i]; j++) {
								pattern_1d[temp++] = PATTERN_SET[i][j];
							}
						}

						/* GPU ������ */

						int* dev_text;
						int* dev_p;
						int* dev_hash_Arr;
						//int* dev_phi_inv;
						int* dev_E;
						int* dev_match;
						int* dev_match_count;

						//********************************** finger �� ��� *******************************************//

						HANDLE_ERROR(hipMalloc((void**)&dev_p, res * sizeof(int)));//pattern
						HANDLE_ERROR(hipMalloc((void**)&dev_text, TEXT_SIZE * sizeof(int)));
						//HANDLE_ERROR(hipMalloc((void**)&dev_p_length, PATTERN_COUNT * sizeof(int)));
						HANDLE_ERROR(hipMalloc((void**)&dev_hash_Arr, PATTERN_COUNT * sizeof(int)));
						//HANDLE_ERROR(hipMalloc((void**)&dev_phi_inv, PATTERN_LEN * PATTERN_COUNT * sizeof(int)));//make 1d arr!
						HANDLE_ERROR(hipMalloc((void**)&dev_E, res * sizeof(int)));
						HANDLE_ERROR(hipMalloc((void**)&dev_match, 5 * 100'000 * sizeof(int)));
						HANDLE_ERROR(hipMalloc((void**)&dev_match_count, 1 * sizeof(int)));

						//copy_stime = clock();
						HANDLE_ERROR(hipMemcpy(dev_p, pattern_1d, res * sizeof(int), hipMemcpyHostToDevice));
						HANDLE_ERROR(hipMemcpy(dev_text, Text, TEXT_SIZE * sizeof(int), hipMemcpyHostToDevice));
						HANDLE_ERROR(hipMemset(dev_match_count, 0, 1 * sizeof(int)));
						HANDLE_ERROR(hipMemset(dev_match, 0, 5 * 100'000 * sizeof(int)));
						// PatternSet�� ��ó���Ͽ� ���������� Ȯ���ϴµ� ���Ǵ� phi_inverse, E ���
						gettimeofday(&PreStart, NULL);
						preprocessing_phi(PATTERN_SET, phi, phi_inv, E, PATTERN_COUNT, PATTERN_LEN);

						// �� ������ ������ q�׷��� ����Ͽ� FingerPrint Table ����
						preprocessing_table(PATTERN_SET, BLOCK_SIZE, PATTERN_COUNT, PATTERN_LEN, hash_Arr, inverse_hash_Arr);
						gettimeofday(&PreEnd, NULL);

						phi_inv_1d = new int[PATTERN_LEN * PATTERN_COUNT];
						E_1d = new int[res];
						temp = 0;
						for (int i = 0; i < PATTERN_COUNT; i++) {
							for (int j = 0; j < PATTERN_LEN; j++) {
								phi_inv_1d[temp++] = phi_inv[i][j];
							}
						}

						temp = 0;
						for (int i = 0; i < PATTERN_COUNT; i++) {
							for (int j = 0; j < pattern_length[i]; j++) {
								E_1d[temp++] = E[i][j];
							}
						}
						//HANDLE_ERROR(hipMemcpy(dev_phi_inv, phi_inv_1d, res * sizeof(int), hipMemcpyHostToDevice));
						HANDLE_ERROR(hipMemcpy(dev_E, E_1d, res * sizeof(int), hipMemcpyHostToDevice));
						HANDLE_ERROR(hipMemcpy(dev_hash_Arr, hash_Arr, PATTERN_COUNT * sizeof(int), hipMemcpyHostToDevice));


						HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_phi_inv), phi_inv_1d, PATTERN_COUNT * PATTERN_LEN * sizeof(int)));

						// ������ ���̺��� Search ����
						gettimeofday(&SearchStart, NULL);
						Search << < ((TEXT_SIZE + 1023) / 1024), 1024 >> > (dev_match_count, dev_match, dev_text, dev_p, dev_hash_Arr, dev_E, PATTERN_COUNT, PATTERN_LEN, BLOCK_SIZE, TEXT_SIZE);
						hipDeviceSynchronize();

						gettimeofday(&SearchEnd, NULL);

						//��ġ�� ����� host�� ����
						match = new int[5 * 100'000];
						match_count = new int[1];
						HANDLE_ERROR(hipMemcpy(match_count, dev_match_count, 1 * sizeof(int), hipMemcpyDeviceToHost));
						HANDLE_ERROR(hipMemcpy(match, dev_match, 5 * 100'000 * sizeof(int), hipMemcpyDeviceToHost));

						int host_match_count = match_count[0];
						OutputData(PATTERN_COUNT, PATTERN_LEN, TEXT_SIZE, BLOCK_SIZE, FolderNumber, host_match_count);

						/*for (int col = 0; col < PATTERN_COUNT; col++) {
								for (int row = 0; row < TEXT_SIZE; row++) {
									out << (match[col*TEXT_SIZE + row]) << " ";
								}
								out << "\n";
						}*/

						gettimeofday(&TotalEnd, NULL);

						hipFree(dev_p);
						hipFree(dev_E);
						hipFree(dev_hash_Arr);
						hipFree(dev_phi_inv);
						hipFree(dev_text);
						hipFree(dev_match_count);
						hipFree(dev_match);

						delete[] match;
						delete[] match_count;
						delete[] pattern_length;
						delete[] pattern_1d;
						delete[] E_1d;
						delete[] phi_inv_1d;
						delete[] hash_Arr;
						delete[] inverse_hash_Arr;

						for (int i = 0; i < PATTERN_COUNT; i++) {
							delete[] phi[i];
							delete[] phi_inv[i];
							delete[] E[i];
						}
						delete[] phi;
						delete[] phi_inv;
						delete[] E;

						for (int i = 0; i < PATTERN_COUNT; i++) {
							delete[] PATTERN_SET[i];
						}
						delete[] Text;
						delete[] PATTERN_SET;

						sec = TotalEnd.tv_sec - TotalStart.tv_sec;
						usec = TotalEnd.tv_usec - TotalStart.tv_usec;
						Total += (sec * 1000 + usec / 1000.0);

						sec = PreEnd.tv_sec - PreStart.tv_sec;
						usec = PreEnd.tv_usec - PreStart.tv_usec;
						TotalPre += (sec * 1000 + usec / 1000.0);

						sec = SearchEnd.tv_sec - SearchStart.tv_sec;
						usec = SearchEnd.tv_usec - SearchStart.tv_usec;
						TotalSearch += (sec * 1000 + usec / 1000.0);

						sec = CopyToHostEnd.tv_sec - CopyToHostStart.tv_sec;
						usec = CopyToHostEnd.tv_usec - CopyToHostStart.tv_usec;
						TotalCopy += (sec * 1000 + usec / 1000.0);
					}
					OutputTime(TotalPre, TotalSearch, Total, TotalCopy, PATTERN_COUNT, PATTERN_LEN, TEXT_SIZE, BLOCK_SIZE);
				}
			}
		}
	}
	cout << endl;
	return 0;
}

//////////////////////////////////////////////////////////////////////////////////////
